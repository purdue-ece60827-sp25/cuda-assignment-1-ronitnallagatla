#include "hip/hip_runtime.h"

#include "cudaLib.cuh"

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void saxpy_gpu(float *x, float *y, float scale, int size) {
  //	Insert GPU SAXPY kernel code here
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < size)
    y[i] += scale * x[i];
}

int runGpuSaxpy(int vectorSize) {

  std::cout << "Hello GPU Saxpy!\n";

  //	Insert code here
  uint64_t size = vectorSize * sizeof(float);

  float *x, *y, *y_init;
  float *x_d, *y_d;

  x = (float *)malloc(size);
  y = (float *)malloc(size);
  y_init = (float *)malloc(size);

  if (x == NULL || y == NULL || y_init == NULL) {
    printf("runGpuSaxpy: unable to malloc memory");
    return -1;
  }

  vectorInit(x, vectorSize);
  vectorInit(y, vectorSize);

  std::memcpy(y_init, y, size);
  float scale = 2.7f;

  hipError_t err;
  err = hipMalloc((void **)&x_d, size);
  err = hipMalloc((void **)&y_d, size);

  if (err != hipSuccess) {
    printf("hipMalloc: unable to malloc memory");
    return -1;
  }

  hipMemcpy(x_d, x, size, hipMemcpyHostToDevice);
  hipMemcpy(y_d, y, size, hipMemcpyHostToDevice);

  // Launch kernel
  // 256 threads in a TB
  saxpy_gpu<<<(vectorSize + 255) / 256, 256>>>(x_d, y_d, scale, vectorSize);

  hipMemcpy(y, y_d, size, hipMemcpyDeviceToHost);

  int error_count = verifyVector(x, y_init, y, scale, vectorSize);
  std::cout << "Found " << error_count << " / " << vectorSize << " errors \n";

  hipFree(x_d);
  hipFree(y_d);
  free(x);
  free(y);

  return 0;
}

/*
 Some helpful definitions

 generateThreadCount is the number of threads spawned initially. Each thread is
 responsible for sampleSize points. *pSums is a pointer to an array that holds
 the number of 'hit' points for each thread. The length of this array is
 pSumSize.

 reduceThreadCount is the number of threads used to reduce the partial sums.
 *totals is a pointer to an array that holds reduced values.
 reduceSize is the number of partial sums that each reduceThreadCount reduces.

*/

__global__ void generatePoints(uint64_t *pSums, uint64_t pSumSize,
                               uint64_t sampleSize) {
  //	Insert code here
}

__global__ void reduceCounts(uint64_t *pSums, uint64_t *totals,
                             uint64_t pSumSize, uint64_t reduceSize) {
  //	Insert code here
}

int runGpuMCPi(uint64_t generateThreadCount, uint64_t sampleSize,
               uint64_t reduceThreadCount, uint64_t reduceSize) {

  //  Check CUDA device presence
  int numDev;
  hipGetDeviceCount(&numDev);
  if (numDev < 1) {
    std::cout << "CUDA device missing!\n";
    return -1;
  }

  auto tStart = std::chrono::high_resolution_clock::now();

  float approxPi = estimatePi(generateThreadCount, sampleSize,
                              reduceThreadCount, reduceSize);

  std::cout << "Estimated Pi = " << approxPi << "\n";

  auto tEnd = std::chrono::high_resolution_clock::now();

  std::chrono::duration<double> time_span = (tEnd - tStart);
  std::cout << "It took " << time_span.count() << " seconds.";

  return 0;
}

double estimatePi(uint64_t generateThreadCount, uint64_t sampleSize,
                  uint64_t reduceThreadCount, uint64_t reduceSize) {

  double approxPi = 0;

  //      Insert code here
  std::cout << "Sneaky, you are ...\n";
  std::cout << "Compute pi, you must!\n";
  return approxPi;
}
